#include "hip/hip_runtime.h"
#include "cuda_main.h"

// === Pipe flow ===
float *h_rad, *h_Q, *h_Rcc;
float *d_rad, *d_Q, *d_Rcc;
int *h_Rc, *h_nRc;
int *d_Rc, *d_nRc;
float *d_R, *d_J, *d_negdQ, *d_residual_NR;

// === CG ===
float *d_x, *d_r, *d_p;
float *d_x_, *d_r_, *d_p_;
float *d_scalars; // p_Ap, r_r old, r_r new, alpha, beta
float *d_Ap, *d_p_A;

void allocate_memory(){
    size_t sz = N * sizeof(float);
    hipError_t err;
    h_a = (float*) malloc(sz);
    err = hipMalloc((void**)&d_a, sz);
    printf("CUDA error (malloc d_a) = %s\n", hipGetErrorString(err));
}
void free_memory(){
    if(h_a) free(h_a);
    if(d_a) hipFree(d_a);
}

void input_from_file(){
    for(int i=0; i<N; ++i)
        h_a[i] = 1.0;
        // h_a[i] = i;
}
void output_to_file(){
    for(int i=0; i<N; ++i)
        printf("%f ", h_a[i]);
    putchar('\n');
    // printf("%f\n", h_a[0]);
}

void send_to_device(){
    size_t sz = N * sizeof(float);
    hipError_t err;

    err = hipMemcpy(d_a, h_a, sz, hipMemcpyHostToDevice);
    printf("CUDA error (memcpy h_a -> d_a) = %s\n", hipGetErrorString(err));
}
void get_from_device(){
    size_t sz = N * sizeof(float);
    hipError_t err;

    err = hipMemcpy(h_a, d_a, sz, hipMemcpyDeviceToHost);
    printf("CUDA error (memcpy d_a -> h_a) = %s\n", hipGetErrorString(err));
}


__global__ void DotProduct(float *x, float *y, float *z, int n){ // z = xTy
    __shared__ float tmp[TPB];  // shared memory useful when data is used serveral times
    int id = blockDim.x*blockIdx.x + threadIdx.x;
    int ii=threadIdx.x;

    tmp[ii] = id<n? x[id]*y[id] : 0.0;
    z[id] = 0.0;
    __syncthreads();
    for(int stride = blockDim.x >> 1; stride > 0; stride >>= 1){ // parallel reduction
        if(ii<stride) tmp[ii] += tmp[ii + stride];             // blockDim.x should be 2^n, otherwise some entry will miss
        __syncthreads();
    }
    if(ii == 0) atomicAdd(&z[0], tmp[0]);
}
__global__ void MatrixVectorProduct(float *A, float *x, float *b, int n){ // Ax=b
	int id = blockDim.x*blockIdx.x + threadIdx.x;

	if(id < n){
		float sum = 0.0;
		for(int i=0; i<n; ++i)	sum += A[id*n+i] * x[i];
		b[id] = sum;
	}
}
__global__ void VectorMatrixProduct(float *x, float *A, float *b, int n){ // ATx=b
	int id = blockDim.x*blockIdx.x + threadIdx.x;

	if(id<n){
		float sum = 0.0;
		for(int i=0; i<n; ++i)	sum += A[i*n+id] * x[i];
		b[id] = sum;
	}
}

__global__ void BiCG_Init(float *b, float *x, float *x_, float *r, float *r_, float *p, float *p_, int n){
	// set both x to zero
	int id = blockDim.x*blockIdx.x + threadIdx.x;
	if(id<n){
		x[id] = 0;	x_[id] = 0;
		r[id] = b[id];	r_[id] = b[id];
		p[id] = b[id];	p_[id] = b[id];
	}
}
__global__ void BiCG_Compute_Alpha(float *scalars){
	scalars[3] = scalars[1]/scalars[0];
}
__global__ void BiCG_Update_x_and_r(float *x, float *x_, float *p, float *p_, float *r, float *r_, float *Ap, float *p_A, float *scalars, int n){
	int id = blockDim.x*blockIdx.x + threadIdx.x;
	if(id<n){
		float alpha = scalars[3];
		x[id] += alpha*p[id];	x_[id] += alpha*p_[id];
		r[id] -= alpha*Ap[id];	r_[id] -= alpha*p_A[id];
	}
}
__global__ void BiCG_Compute_Beta(float *scalars){
	scalars[4] = scalars[2]/scalars[1];
}
__global__ void BiCG_Update_p(float *p, float *p_, float *r, float *r_, float *scalars, int n){
	int id = blockDim.x*blockIdx.x + threadIdx.x;
	if(id<n){
		float beta = scalars[4];
		p[id] = r[id] + beta*p[id];
		p_[id] = r_[id] + beta*p_[id];
	}
}
__global__ void BiCG_Update_residual(float *scalars){
	scalars[1] = scalars[2];
}
void BiCG(float *A, float *x, float *b, int n){
	int i;
	float residual; // RTR

	BiCG_Init<<<BPG,TPB>>>(b, x, d_x_, d_r, d_r_, d_p, d_p_, n);
	DotProduct<<<BPG,TPB>>>(d_r, d_r, d_scalars+1, n, d_tmp_BPG);
	for(i=0; i<ITER_CG; ++i){
		MatrixVectorProduct<<<BPG,TPB>>>(A, d_p, d_Ap, n);
		VectorMatrixProduct<<<BPG,TPB>>>(d_p_, A, d_p_A, n);
		DotProduct<<<BPG,TPB>>>(d_p_, d_Ap, d_scalars, n, d_tmp_BPG);
		BiCG_Compute_Alpha<<<1,1>>>(d_scalars);
		BiCG_Update_x_and_r<<<BPG,TPB>>>(x, d_x_, d_p, d_p_, d_r, d_r_, d_Ap, d_p_A, d_scalars, n);
		DotProduct<<<BPG,TPB>>>(d_r_, d_r, d_scalars+2, n, d_tmp_BPG); // RTR new
		hipMemcpy(&residual, d_scalars+2, sizeof(float), hipMemcpyDeviceToHost);
		if(fabsf(residual) < EPS_CG)	break;
		BiCG_Compute_Beta<<<1,1>>>(d_scalars);
		BiCG_Update_p<<<BPG,TPB>>>(d_p, d_p_, d_r, d_r_, d_scalars, n);
		BiCG_Update_residual<<<1,1>>>(d_scalars);
	}
	//printf("i=%d\n", i);
}

// Pipe flow
__global__ void Compute_R_and_J(float *rad, float *Q, float *Rcc, int *Rc, int *nRc, float *R, float *J, int n){
	int id = blockDim.x*blockIdx.x + threadIdx.x;
	int i, tmp, offset = id*n;
	float sum, tmpQ;

	if(id < N_LOOPS){ // loop eqns
		sum = Rcc[id];
		for(i=0; i<n; ++i)	J[offset+i] = 0;
		for(i=0; i<nRc[id]; ++i){
			tmp = Rc[offset+i];
			if(tmp < 0){
				tmp = -tmp-1;
				tmpQ = Q[tmp];
				sum -= rad[tmp]*fabsf(tmpQ)*tmpQ; // R
				J[offset+tmp] = -2*rad[tmp]*tmpQ; // J
			}
			else{
				tmp = tmp-1;
				tmpQ = Q[tmp];
				sum += rad[tmp]*fabsf(tmpQ)*tmpQ;
				J[offset+tmp] = 2*rad[tmp]*tmpQ;
			}
		}
		R[id] = sum;
	}
	else if(id < n){ // nodes eqns
		sum = Rcc[id];
		for(i=0; i<n; ++i)	J[offset+i] = 0;
		for(i=0; i<nRc[id]; ++i){
			tmp = Rc[offset+i];
			if(tmp < 0){
				tmp = -tmp-1;
				sum -= Q[tmp]; // R
				J[offset+tmp] = -1.0; // J
			}
			else{
				tmp = tmp-1;
				sum += Q[tmp];
				J[offset+tmp] = 1.0;
			}
		}
		R[id] = sum;
	}
}
__global__ void Update_Q(float *Q, float *negdQ, int n){
	int id = blockDim.x*blockIdx.x + threadIdx.x;
	if(id<n){
		Q[id] -= DAMP_NR*negdQ[id];
	}
}

void Newton_Raphson(float *R, float *J, float *Q, int n){ // find the sol of R=0, with NR use J*dx=-R, x=x+dx => J*(-dx)=R, x=x-(-dx)
	int i;
	float residual;

	for(i=0; i<ITER_NR; ++i){
		Compute_R_and_J<<<BPG,TPB>>>(d_rad, Q, d_Rcc, d_Rc, d_nRc, R, J, n);
		DotProduct<<<BPG,TPB>>>(R, R, d_residual_NR, n, d_tmp_BPG);
		hipMemcpy(&residual, d_tmp_BPG, sizeof(float), hipMemcpyDeviceToHost);
		if(residual < EPS_NR)	break;
		BiCG(J, d_negdQ, R, n);
		Update_Q<<<BPG,TPB>>>(Q, d_negdQ, n);
	}

	printf("iteration_NR = %d\n", i);
	printf("residual_NR = %.10e\n", residual);
}

// https://www.cs.cmu.edu/afs/cs/academic/class/15668-s11/www/cuda-doc/html/group__CUDART__DEVICE_g5aa4f47938af8276f08074d09b7d520c.html
// https://zhuanlan.zhihu.com/p/41151532


