#include "hip/hip_runtime.h"
#include "hydra_cuda.h"
#include <hiprand.h>

// https://docs.nvidia.com/cuda/cuda-math-api/index.html
// https://docs.nvidia.com/cuda/cusolver/index.html

// gpu kernels for bicg
void all_within(int n, double *x, double lowerb, double upperb){
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    // do this outside? no good way to do this in GPU?
}
__global__ void rands(int n, double *x, double a = 1.0, double b = 0.0){
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    // https://stackoverflow.com/questions/22425283/how-could-we-generate-random-numbers-in-cuda-c-with-different-seed-on-each-run
    // https://developer.nvidia.com/hiprand
}
__global__ void x_plus_sy(int n, double *x, double s, double *y, double *z){ // minus by -s
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if(id < n)
        z[id] = x[id] + s * y[id];
}
__global__ void Ax(int n, double *A, double *x, double *y){ // y = Ax
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if(id < n){
        double tmp = 0.0;
        for(int i = 0; i < n; ++i)
            tmp += A[id*n+i] * x[i];
        y[id] = tmp;
    }
}
__global__ void ATx(int n, double *A, double *x, double *y){ // y = ATx
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if(id < n){
        double tmp = 0.0;
        for(int i = 0; i < n; ++i)
            tmp += A[i*n+id] * x[i];
        y[id] = tmp;
    }
}
__global__ void xTy(int n, float *x, float *y, float *z){ // z = xTy
    __shared__ float tmp[TPB];  // shared memory useful when data is used serveral times
    int id = blockDim.x*blockIdx.x + threadIdx.x;
    int ii = threadIdx.x;

    tmp[ii] = id<n? x[id]*y[id] : 0.0;
    z[id] = 0.0;
    __syncthreads();
    for(int stride = blockDim.x >> 1; stride > 0; stride >>= 1){ // parallel reduction
        if(ii < stride) tmp[ii] += tmp[ii + stride]; // blockDim.x should be 2^n, otherwise some entries will miss
        __syncthreads();
    }
    if(ii == 0) z[blockIdx.x] = tmp[0];
    __syncthreads();
    if(id == 0){
        float sum = 0.0;
        for(int i = 0; i < gridDim.x; i += 2) sum += z[i];
        z[0] = sum;
    }
    else if(id == 1){
        float sum = 0.0;
        for(int i = 1; i < gridDim.x; i += 2) sum += z[i];
        z[1] = sum;
    }
    __syncthreads();
    if(id == 0) z[0] += z[1];
}
void bicg(int n, double *A, double *x, double *b, int maxattempts = 100, int maxiters = 1000, double tol = 1e-6){
    int i, j, k;

    if(n * 10 > maxiters) maxiters = n * 10;
    // malloc in solve_cuda
    // try different initial guess several times
    for(i = 0; i < maxattemps; ++i){
        // init
        // rands(n, x, n * sizeof(double));
        // ...
        // start iteration
        for(j = 0; j < maxiters; ++j){
            // ...

            // don't check residual every iteration
            // for example, check every 100 iteration

            // ...
        }
        // check

    }
    // free in solve_cuda
}


__global__ void compute_R_and_J(){
    int id = blockDim.x*blockIdx.x + threadIdx.x;
    // compute R

    // compute J
}
void solve_cuda(int nE, int nLeq, int nNeq, double *incLoop, double *conLoop, double *incNode, double *conNode, double *&x, double n, int maxiters, int maxattempts, double tol, double step){
    int i, j;

    // allocate memory

    // send to device

    // newton's method
    for(i = 0; i < maxattempts; ++i){
        // init cpu vaiables

        // init gpu variables

        // start iterations
        for(j = 0; j < maxiters; ++j){


        }
        // check
    }

    // get from device

    // free_memory

}
